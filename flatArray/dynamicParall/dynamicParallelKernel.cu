#include "hip/hip_runtime.h"
#include "dynamicParallelKernel.h"

namespace FADPKERNEL{

FaDPSolver::FaDPSolver(tValue* dv_ptr, tSize size):dv_ptr(dv_ptr), size(size){

}

void FaDPSolver::solve(){

}

__global__ void dynamicParallelKernel(tValue* dv_ptr, tSize size, tSize step_size){

}


__global__ void __launch_bounds__(1024)
dynamicParallelKernelChild(tValue* dst_ptr, tValue* src_ptr, tSize size){
	const int threadId = threadIdx.x;
	const int threadNum = blockDim.x;
	for(int i = threadId; i < size; i+= threadNum){
		dst_ptr[i] += src_ptr[i];
	}
	return;
}

};
