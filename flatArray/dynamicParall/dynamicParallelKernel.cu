#include "hip/hip_runtime.h"
#include "dynamicParallelKernel.h"

namespace FADPKERNEL{

FaDPSolver::FaDPSolver(tValue* dv_ptr, tSize size):dv_ptr(dv_ptr), size(size){

}

void FaDPSolver::solve(){
	dynamicParallelKernel<<<1, 4>>>(dv_ptr, size, STEP);
	gpuErrchk( hipDeviceSynchronize() );
}

//step size shouldn't be too small. One child kernel process step size values.
__global__ void __launch_bounds__(32)
dynamicParallelKernel(tValue* dv_ptr, tSize size, tSize step_size){
	const int threadId = threadIdx.x;
	const int threadNum = blockDim.x;

	tValue* dst_ptr;
	tValue* src_ptr;
	tSize child_size = 0;
	
	int num_launch1 = size/(2*step_size);

	for(int i = threadId; i < num_launch1; i += threadNum){
		dst_ptr = dv_ptr + 2*i*step_size;
		src_ptr = dst_ptr + step_size;
		if((2*i + 1) * step_size > size){
			child_size = size - 2*i*step_size;
		}else {
			child_size = step_size;
		}

		dynamicParallelKernelChild<<<1, 512>>>(dst_ptr, src_ptr, child_size);
	}
	hipDeviceSynchronize();
	__syncthreads();

	int num_launch2 = (size - step_size)/(2*step_size);

	for(int i = threadId; i < num_launch2 ; i += threadNum){
		dst_ptr = dv_ptr + step_size + 2*i*step_size;
		src_ptr = dst_ptr + step_size;
		if((2*i + 1) * step_size > size - step_size){
			child_size = size - step_size - 2*i*step_size;
		}else {
			child_size = step_size;
		}
		dynamicParallelKernelChild<<<1, 512>>>(dst_ptr, src_ptr, child_size);
	}
	hipDeviceSynchronize();
	__syncthreads();
}


__global__ void __launch_bounds__(1024)
dynamicParallelKernelChild(tValue* dst_ptr, tValue* src_ptr, tSize size){
	const int threadId = threadIdx.x;
	const int threadNum = blockDim.x;
	for(int i = threadId; i < size; i+= threadNum){
		dst_ptr[i] += src_ptr[i];
	}
	__syncthreads();
	return;
}

};
