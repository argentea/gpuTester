#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "database.h"
#include "utils.h"

namespace FLATARRAY{

void FlatArray::dinit(){
	gpuErrchk(hipMalloc((void **)&dv_ptr, size*sizeof(tValue)));
	std::cout << size << std::endl;
	incSegGen<<<4, 256>>>(dv_ptr, size, STEP);
	gpuErrchk( hipDeviceSynchronize() );
}

void FlatArray::dToH(){
	gpuErrchk(hipMemcpy(hv_ptr, dv_ptr, size*sizeof(tValue), hipMemcpyDeviceToHost));
}

void FlatArray::hTod(){
	gpuErrchk(hipMemcpy(dv_ptr, hv_ptr, size*sizeof(tValue), hipMemcpyHostToDevice));
}


};
