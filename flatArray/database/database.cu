#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "database.h"
#include "utils.h"

namespace FLATARRAY{

void FlatArray::dinit(tSize isize){
	gpuErrchk(hipMalloc((void **)&dv_ptr, isize*sizeof(tSize)));
}

};
