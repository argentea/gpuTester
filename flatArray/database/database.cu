#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "database.h"
#include "utils.h"

namespace FLATARRAY{

void FlatArray::dinit(){
	gpuErrchk(hipMalloc((void **)&dv_ptr, size*sizeof(tValue)));
}

void FlatArray::dToH(){
	gpuErrchk(hipMemcpy(hv_ptr, dv_ptr, size*sizeof(tValue), hipMemcpyDeviceToHost));
}

void FlatArray::hTod(){
	gpuErrchk(hipMemcpy(dv_ptr, hv_ptr, size*sizeof(tValue), hipMemcpyHostToDevice));
}


};
