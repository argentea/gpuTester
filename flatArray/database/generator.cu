#include "hip/hip_runtime.h"
#include "generator.h"

__global__ void __launch_bounds__(1024)
incSegGen(tValue* dv_ptr, tSize size, tSize step){
	const int threadId = threadIdx.x;
	const int threadNum = blockDim.x;
	const int blockId = blockIdx.x;
	const int blockNum = gridDim.x;


	int tsize = step;
	int bucket_num = size/step + 1;
	for(int i = blockId; i < bucket_num; i += blockNum){
		if(step * (i + 1) > size){
			tsize = size - step*i;
		}
		for(int j = threadId; j < tsize; j += threadNum){
			dv_ptr[i*step + j] = i;
		}
	}
	__syncthreads();
}
